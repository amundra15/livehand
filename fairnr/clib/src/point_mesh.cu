#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All rights reserved.

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <algorithm>
#include <list>
#include <queue>
#include <tuple>
#include "float_math.cuh"
#include "geometry_utils.cuh"
#include "warp_reduce.cuh"

// ****************************************************************************
// *                          PointFaceDistance                               *
// ****************************************************************************

__global__ void PointFaceForwardKernel(
    const float* __restrict__ points, // (P, 3)
    const int64_t* __restrict__ points_first_idx, // (B,)
    const float* __restrict__ tris, // (T, 3, 3)
    const int64_t* __restrict__ tris_first_idx, // (B,)
    float* __restrict__ dist_points, // (P,)
    int64_t* __restrict__ idx_points, // (P,)
    float* __restrict__ w0, // (P,)
    float* __restrict__ w1, // (P,)
    float* __restrict__ w2, // (P,)
    const size_t B,
    const size_t P,
    const size_t T) {
        
  float3* points_f3 = (float3*)points;
  float3* tris_f3 = (float3*)tris;

  // Single shared memory buffer which is split and cast to different types.
  extern __shared__ char shared_buf[];

  float* min_dists = (float*)shared_buf; // float[NUM_THREADS]
  float* mw0 = (float*)&min_dists[blockDim.x];
  float* mw1 = (float*)&min_dists[blockDim.x * 2];
  float* mw2 = (float*)&min_dists[blockDim.x * 3];
  int64_t* min_idxs = (int64_t*)&min_dists[blockDim.x * 4]; // int64_t[NUM_THREADS]
  
  const size_t batch_idx = blockIdx.y; // index of batch element.

  // start and end for points in batch_idx
  const int64_t startp = points_first_idx[batch_idx];
  const int64_t endp = batch_idx + 1 < B ? points_first_idx[batch_idx + 1] : P;

  // start and end for faces in batch_idx
  const int64_t startt = tris_first_idx[batch_idx];
  const int64_t endt = batch_idx + 1 < B ? tris_first_idx[batch_idx + 1] : T;

  const size_t i = blockIdx.x; // index of point within batch element.
  const size_t tid = threadIdx.x; // thread index

  // Each block will compute one element of the output idx_points[startp + i],
  // dist_points[startp + i]. Within the block we will use threads to compute
  // the distances between points[startp + i] and tris[j] for all j belonging
  // in the same batch as i, i.e. j in [startt, endt]. Then use a block
  // reduction to take an argmin of the distances.

  // If i exceeds the number of points in batch_idx, then do nothing
  if (i < (endp - startp)) {
    // Retrieve (startp + i) point
    const float3 p_f3 = points_f3[startp + i];

    // Compute the distances between points[startp + i] and tris[j] for
    // all j belonging in the same batch as i, i.e. j in [startt, endt].
    // Here each thread will reduce over (endt-startt) / blockDim.x in serial,
    // and store its result to shared memory
    float min_dist = FLT_MAX;
    size_t min_idx = 0;
    float min_w0 = -1;
    float min_w1 = -1;
    float min_w2 = -1;

    for (size_t j = tid; j < (endt - startt); j += blockDim.x) {
      const float3 v0 = tris_f3[(startt + j) * 3 + 0];
      const float3 v1 = tris_f3[(startt + j) * 3 + 1];
      const float3 v2 = tris_f3[(startt + j) * 3 + 2];
      thrust::tuple<float, float3> outs = PointTriangle3DistanceCoordsForward(p_f3, v0, v1, v2);
      float  dist = thrust::get<0>(outs);
      float3 bary = thrust::get<1>(outs);
      // printf("%f : %f %f %f %d |\n", dist, bary.x, bary.y, bary.z, startt + j);
      min_dist = (j == tid) ? dist : min_dist;
      min_idx = (dist <= min_dist) ? (startt + j) : min_idx;
      min_w0 = (dist <= min_dist) ? bary.x : min_w0;
      min_w1 = (dist <= min_dist) ? bary.y : min_w1;
      min_w2 = (dist <= min_dist) ? bary.z : min_w2;
      min_dist = (dist <= min_dist) ? dist : min_dist;
    }
    min_dists[tid] = min_dist;
    min_idxs[tid] = min_idx;
    mw0[tid] = min_w0;
    mw1[tid] = min_w1;
    mw2[tid] = min_w2;
    __syncthreads();

    // Perform reduction in shared memory.
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
      if (tid < s) {
        if (min_dists[tid] > min_dists[tid + s]) {
          min_dists[tid] = min_dists[tid + s];
          min_idxs[tid] = min_idxs[tid + s];
          mw0[tid] = mw0[tid + s];
          mw1[tid] = mw1[tid + s];
          mw2[tid] = mw2[tid + s];
        }
      }
      __syncthreads();
    }

    // Unroll the last 6 iterations of the loop since they will happen
    // synchronized within a single warp.
    if (tid < 32)
      WarpReduce2<float>(min_dists, min_idxs, mw0, mw1, mw2, tid);

    // Finally thread 0 writes the result to the output buffer.
    if (tid == 0) {
      idx_points[startp + i] = min_idxs[0];
      dist_points[startp + i] = min_dists[0];
      w0[startp + i] = mw0[0];
      w1[startp + i] = mw1[0];
      w2[startp + i] = mw2[0];
    }
  }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor, at::Tensor> 
PointFaceDistanceForwardCuda(
    const at::Tensor& points,
    const at::Tensor& points_first_idx,
    const at::Tensor& tris,
    const at::Tensor& tris_first_idx,
    const int64_t max_points) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      points_first_idx_t{points_first_idx, "points_first_idx", 2},
      tris_t{tris, "tris", 3},
      tris_first_idx_t{tris_first_idx, "tris_first_idx", 4};
  at::CheckedFrom c = "PointFaceDistanceForwardCuda";
  at::checkAllSameGPU(
      c, {points_t, points_first_idx_t, tris_t, tris_first_idx_t});
  at::checkAllSameType(c, {points_t, tris_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int64_t P = points.size(0);
  const int64_t T = tris.size(0);
  const int64_t B = points_first_idx.size(0);

  TORCH_CHECK(points.size(1) == 3, "points must be of shape Px3");
  TORCH_CHECK(
      (tris.size(1) == 3) && (tris.size(2) == 3),
      "tris must be of shape Tx3x3");
  TORCH_CHECK(tris_first_idx.size(0) == B);

  // clang-format off
  at::Tensor dists = at::zeros({P,}, points.options());
  at::Tensor w0 = at::zeros({P,}, points.options());
  at::Tensor w1 = at::zeros({P,}, points.options());
  at::Tensor w2 = at::zeros({P,}, points.options());
  at::Tensor idxs = at::zeros({P,}, points_first_idx.options());
  // clang-format on

  if (dists.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(dists, idxs, w0, w1, w2);
  }

  const int threads = 128;
  const dim3 blocks(max_points, B);
  size_t shared_size = 4 * threads * sizeof(size_t) + threads * sizeof(int64_t);

  PointFaceForwardKernel<<<blocks, threads, shared_size, stream>>>(
      points.data_ptr<float>(),
      points_first_idx.data_ptr<int64_t>(),
      tris.data_ptr<float>(),
      tris_first_idx.data_ptr<int64_t>(),
      dists.data_ptr<float>(),
      idxs.data_ptr<int64_t>(),
      w0.data_ptr<float>(),
      w1.data_ptr<float>(),
      w2.data_ptr<float>(),
      B,
      P,
      T);

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(dists, idxs, w0, w1, w2);
}

__global__ void PointFaceBackwardKernel(
    const float* __restrict__ points, // (P, 3)
    const float* __restrict__ tris, // (T, 3, 3)
    const int64_t* __restrict__ idx_points, // (P,)
    const float* __restrict__ grad_dists, // (P,)
    float* __restrict__ grad_points, // (P, 3)
    float* __restrict__ grad_tris, // (T, 3, 3)
    const size_t P) {
  float3* points_f3 = (float3*)points;
  float3* tris_f3 = (float3*)tris;

  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;

  for (size_t p = tid; p < P; p += stride) {
    const float3 p_f3 = points_f3[p];

    const int64_t tidx = idx_points[p];
    const float3 v0 = tris_f3[tidx * 3 + 0];
    const float3 v1 = tris_f3[tidx * 3 + 1];
    const float3 v2 = tris_f3[tidx * 3 + 2];

    const float grad_dist = grad_dists[p];

    const auto grads =
        PointTriangle3DistanceBackward(p_f3, v0, v1, v2, grad_dist);
    const float3 grad_point = thrust::get<0>(grads);
    const float3 grad_v0 = thrust::get<1>(grads);
    const float3 grad_v1 = thrust::get<2>(grads);
    const float3 grad_v2 = thrust::get<3>(grads);

    atomicAdd(grad_points + p * 3 + 0, grad_point.x);
    atomicAdd(grad_points + p * 3 + 1, grad_point.y);
    atomicAdd(grad_points + p * 3 + 2, grad_point.z);

    atomicAdd(grad_tris + tidx * 3 * 3 + 0 * 3 + 0, grad_v0.x);
    atomicAdd(grad_tris + tidx * 3 * 3 + 0 * 3 + 1, grad_v0.y);
    atomicAdd(grad_tris + tidx * 3 * 3 + 0 * 3 + 2, grad_v0.z);

    atomicAdd(grad_tris + tidx * 3 * 3 + 1 * 3 + 0, grad_v1.x);
    atomicAdd(grad_tris + tidx * 3 * 3 + 1 * 3 + 1, grad_v1.y);
    atomicAdd(grad_tris + tidx * 3 * 3 + 1 * 3 + 2, grad_v1.z);

    atomicAdd(grad_tris + tidx * 3 * 3 + 2 * 3 + 0, grad_v2.x);
    atomicAdd(grad_tris + tidx * 3 * 3 + 2 * 3 + 1, grad_v2.y);
    atomicAdd(grad_tris + tidx * 3 * 3 + 2 * 3 + 2, grad_v2.z);
  }
}

std::tuple<at::Tensor, at::Tensor> PointFaceDistanceBackwardCuda(
    const at::Tensor& points,
    const at::Tensor& tris,
    const at::Tensor& idx_points,
    const at::Tensor& grad_dists) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      idx_points_t{idx_points, "idx_points", 2}, tris_t{tris, "tris", 3},
      grad_dists_t{grad_dists, "grad_dists", 4};
  at::CheckedFrom c = "PointFaceDistanceBackwardCuda";
  at::checkAllSameGPU(c, {points_t, idx_points_t, tris_t, grad_dists_t});
  at::checkAllSameType(c, {points_t, tris_t, grad_dists_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int64_t P = points.size(0);
  const int64_t T = tris.size(0);

  TORCH_CHECK(points.size(1) == 3, "points must be of shape Px3");
  TORCH_CHECK(
      (tris.size(1) == 3) && (tris.size(2) == 3),
      "tris must be of shape Tx3x3");
  TORCH_CHECK(idx_points.size(0) == P);
  TORCH_CHECK(grad_dists.size(0) == P);

  // clang-format off
  at::Tensor grad_points = at::zeros({P, 3}, points.options());
  at::Tensor grad_tris = at::zeros({T, 3, 3}, tris.options());
  // clang-format on

  if (grad_points.numel() == 0 || grad_tris.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_points, grad_tris);
  }

  const int blocks = 64;
  const int threads = 512;

  PointFaceBackwardKernel<<<blocks, threads, 0, stream>>>(
      points.data_ptr<float>(),
      tris.data_ptr<float>(),
      idx_points.data_ptr<int64_t>(),
      grad_dists.data_ptr<float>(),
      grad_points.data_ptr<float>(),
      grad_tris.data_ptr<float>(),
      P);

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(grad_points, grad_tris);
}

// ****************************************************************************
// *                          FacePointDistance                               *
// ****************************************************************************

__global__ void FacePointForwardKernel(
    const float* __restrict__ points, // (P, 3)
    const int64_t* __restrict__ points_first_idx, // (B,)
    const float* __restrict__ tris, // (T, 3, 3)
    const int64_t* __restrict__ tris_first_idx, // (B,)
    float* __restrict__ dist_tris, // (T,)
    int64_t* __restrict__ idx_tris, // (T,)
    const size_t B,
    const size_t P,
    const size_t T) {
  float3* points_f3 = (float3*)points;
  float3* tris_f3 = (float3*)tris;

  // Single shared memory buffer which is split and cast to different types.
  extern __shared__ char shared_buf[];
  float* min_dists = (float*)shared_buf; // float[NUM_THREADS]
  int64_t* min_idxs = (int64_t*)&min_dists[blockDim.x]; // int64_t[NUM_THREADS]

  const size_t batch_idx = blockIdx.y; // index of batch element.

  // start and end for points in batch_idx
  const int64_t startp = points_first_idx[batch_idx];
  const int64_t endp = batch_idx + 1 < B ? points_first_idx[batch_idx + 1] : P;

  // start and end for tris in batch_idx
  const int64_t startt = tris_first_idx[batch_idx];
  const int64_t endt = batch_idx + 1 < B ? tris_first_idx[batch_idx + 1] : T;

  const size_t i = blockIdx.x; // index of point within batch element.
  const size_t tid = threadIdx.x;

  // Each block will compute one element of the output idx_tris[startt + i],
  // dist_tris[startt + i]. Within the block we will use threads to compute
  // the distances between tris[startt + i] and points[j] for all j belonging
  // in the same batch as i, i.e. j in [startp, endp]. Then use a block
  // reduction to take an argmin of the distances.

  // If i exceeds the number of tris in batch_idx, then do nothing
  if (i < (endt - startt)) {
    const float3 v0 = tris_f3[(startt + i) * 3 + 0];
    const float3 v1 = tris_f3[(startt + i) * 3 + 1];
    const float3 v2 = tris_f3[(startt + i) * 3 + 2];

    // Compute the distances between tris[startt + i] and points[j] for
    // all j belonging in the same batch as i, i.e. j in [startp, endp].
    // Here each thread will reduce over (endp-startp) / blockDim.x in serial,
    // and store its result to shared memory
    float min_dist = FLT_MAX;
    size_t min_idx = 0;
    for (size_t j = tid; j < (endp - startp); j += blockDim.x) {
      // Retrieve (startp + i) point
      const float3 p_f3 = points_f3[startp + j];

      float dist = PointTriangle3DistanceForward(p_f3, v0, v1, v2);
      min_dist = (j == tid) ? dist : min_dist;
      min_idx = (dist <= min_dist) ? (startp + j) : min_idx;
      min_dist = (dist <= min_dist) ? dist : min_dist;
    }
    min_dists[tid] = min_dist;
    min_idxs[tid] = min_idx;
    __syncthreads();

    // Perform reduction in shared memory.
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
      if (tid < s) {
        if (min_dists[tid] > min_dists[tid + s]) {
          min_dists[tid] = min_dists[tid + s];
          min_idxs[tid] = min_idxs[tid + s];
        }
      }
      __syncthreads();
    }

    // Unroll the last 6 iterations of the loop since they will happen
    // synchronized within a single warp.
    if (tid < 32)
      WarpReduce<float>(min_dists, min_idxs, tid);

    // Finally thread 0 writes the result to the output buffer.
    if (tid == 0) {
      idx_tris[startt + i] = min_idxs[0];
      dist_tris[startt + i] = min_dists[0];
    }
  }
}

std::tuple<at::Tensor, at::Tensor> FacePointDistanceForwardCuda(
    const at::Tensor& points,
    const at::Tensor& points_first_idx,
    const at::Tensor& tris,
    const at::Tensor& tris_first_idx,
    const int64_t max_tris) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      points_first_idx_t{points_first_idx, "points_first_idx", 2},
      tris_t{tris, "tris", 3},
      tris_first_idx_t{tris_first_idx, "tris_first_idx", 4};
  at::CheckedFrom c = "FacePointDistanceForwardCuda";
  at::checkAllSameGPU(
      c, {points_t, points_first_idx_t, tris_t, tris_first_idx_t});
  at::checkAllSameType(c, {points_t, tris_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int64_t P = points.size(0);
  const int64_t T = tris.size(0);
  const int64_t B = points_first_idx.size(0);

  TORCH_CHECK(points.size(1) == 3, "points must be of shape Px3");
  TORCH_CHECK(
      (tris.size(1) == 3) && (tris.size(2) == 3),
      "tris must be of shape Tx3x3");
  TORCH_CHECK(tris_first_idx.size(0) == B);

  // clang-format off
  at::Tensor dists = at::zeros({T,}, tris.options());
  at::Tensor idxs = at::zeros({T,}, tris_first_idx.options());
  // clang-format on

  if (dists.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(dists, idxs);
  }

  const int threads = 128;
  const dim3 blocks(max_tris, B);
  size_t shared_size = threads * sizeof(size_t) + threads * sizeof(int64_t);

  FacePointForwardKernel<<<blocks, threads, shared_size, stream>>>(
      points.data_ptr<float>(),
      points_first_idx.data_ptr<int64_t>(),
      tris.data_ptr<float>(),
      tris_first_idx.data_ptr<int64_t>(),
      dists.data_ptr<float>(),
      idxs.data_ptr<int64_t>(),
      B,
      P,
      T);

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(dists, idxs);
}

__global__ void FacePointBackwardKernel(
    const float* __restrict__ points, // (P, 3)
    const float* __restrict__ tris, // (T, 3, 3)
    const int64_t* __restrict__ idx_tris, // (T,)
    const float* __restrict__ grad_dists, // (T,)
    float* __restrict__ grad_points, // (P, 3)
    float* __restrict__ grad_tris, // (T, 3, 3)
    const size_t T) {
  float3* points_f3 = (float3*)points;
  float3* tris_f3 = (float3*)tris;

  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;

  for (size_t t = tid; t < T; t += stride) {
    const float3 v0 = tris_f3[t * 3 + 0];
    const float3 v1 = tris_f3[t * 3 + 1];
    const float3 v2 = tris_f3[t * 3 + 2];

    const int64_t pidx = idx_tris[t];

    const float3 p_f3 = points_f3[pidx];

    const float grad_dist = grad_dists[t];

    const auto grads =
        PointTriangle3DistanceBackward(p_f3, v0, v1, v2, grad_dist);
    const float3 grad_point = thrust::get<0>(grads);
    const float3 grad_v0 = thrust::get<1>(grads);
    const float3 grad_v1 = thrust::get<2>(grads);
    const float3 grad_v2 = thrust::get<3>(grads);

    atomicAdd(grad_points + pidx * 3 + 0, grad_point.x);
    atomicAdd(grad_points + pidx * 3 + 1, grad_point.y);
    atomicAdd(grad_points + pidx * 3 + 2, grad_point.z);

    atomicAdd(grad_tris + t * 3 * 3 + 0 * 3 + 0, grad_v0.x);
    atomicAdd(grad_tris + t * 3 * 3 + 0 * 3 + 1, grad_v0.y);
    atomicAdd(grad_tris + t * 3 * 3 + 0 * 3 + 2, grad_v0.z);

    atomicAdd(grad_tris + t * 3 * 3 + 1 * 3 + 0, grad_v1.x);
    atomicAdd(grad_tris + t * 3 * 3 + 1 * 3 + 1, grad_v1.y);
    atomicAdd(grad_tris + t * 3 * 3 + 1 * 3 + 2, grad_v1.z);

    atomicAdd(grad_tris + t * 3 * 3 + 2 * 3 + 0, grad_v2.x);
    atomicAdd(grad_tris + t * 3 * 3 + 2 * 3 + 1, grad_v2.y);
    atomicAdd(grad_tris + t * 3 * 3 + 2 * 3 + 2, grad_v2.z);
  }
}

std::tuple<at::Tensor, at::Tensor> FacePointDistanceBackwardCuda(
    const at::Tensor& points,
    const at::Tensor& tris,
    const at::Tensor& idx_tris,
    const at::Tensor& grad_dists) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      idx_tris_t{idx_tris, "idx_tris", 2}, tris_t{tris, "tris", 3},
      grad_dists_t{grad_dists, "grad_dists", 4};
  at::CheckedFrom c = "FacePointDistanceBackwardCuda";
  at::checkAllSameGPU(c, {points_t, idx_tris_t, tris_t, grad_dists_t});
  at::checkAllSameType(c, {points_t, tris_t, grad_dists_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int64_t P = points.size(0);
  const int64_t T = tris.size(0);

  TORCH_CHECK(points.size(1) == 3, "points must be of shape Px3");
  TORCH_CHECK(
      (tris.size(1) == 3) && (tris.size(2) == 3),
      "tris must be of shape Tx3x3");
  TORCH_CHECK(idx_tris.size(0) == T);
  TORCH_CHECK(grad_dists.size(0) == T);

  // clang-format off
  at::Tensor grad_points = at::zeros({P, 3}, points.options());
  at::Tensor grad_tris = at::zeros({T, 3, 3}, tris.options());
  // clang-format on

  if (grad_points.numel() == 0 || grad_tris.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_points, grad_tris);
  }

  const int blocks = 64;
  const int threads = 512;

  FacePointBackwardKernel<<<blocks, threads, 0, stream>>>(
      points.data_ptr<float>(),
      tris.data_ptr<float>(),
      idx_tris.data_ptr<int64_t>(),
      grad_dists.data_ptr<float>(),
      grad_points.data_ptr<float>(),
      grad_tris.data_ptr<float>(),
      T);

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(grad_points, grad_tris);
}

// ****************************************************************************
// *                     PointFaceArrayDistance                               *
// ****************************************************************************

__global__ void PointFaceArrayForwardKernel(
    const float* __restrict__ points, // (P, 3)
    const float* __restrict__ tris, // (T, 3, 3)
    float* __restrict__ dists, // (P, T)
    const size_t P,
    const size_t T) {
  const float3* points_f3 = (float3*)points;
  const float3* tris_f3 = (float3*)tris;

  // Parallelize over P * S computations
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int t_i = tid; t_i < P * T; t_i += num_threads) {
    const int t = t_i / P; // segment index.
    const int p = t_i % P; // point index
    const float3 v0 = tris_f3[t * 3 + 0];
    const float3 v1 = tris_f3[t * 3 + 1];
    const float3 v2 = tris_f3[t * 3 + 2];

    const float3 point = points_f3[p];
    float dist = PointTriangle3DistanceForward(point, v0, v1, v2);
    dists[p * T + t] = dist;
  }
}

at::Tensor PointFaceArrayDistanceForwardCuda(
    const at::Tensor& points,
    const at::Tensor& tris) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1}, tris_t{tris, "tris", 2};
  at::CheckedFrom c = "PointFaceArrayDistanceForwardCuda";
  at::checkAllSameGPU(c, {points_t, tris_t});
  at::checkAllSameType(c, {points_t, tris_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int64_t P = points.size(0);
  const int64_t T = tris.size(0);

  TORCH_CHECK(points.size(1) == 3, "points must be of shape Px3");
  TORCH_CHECK(
      (tris.size(1) == 3) && (tris.size(2) == 3),
      "tris must be of shape Tx3x3");

  at::Tensor dists = at::zeros({P, T}, points.options());

  if (dists.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return dists;
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  PointFaceArrayForwardKernel<<<blocks, threads, 0, stream>>>(
      points.data_ptr<float>(),
      tris.data_ptr<float>(),
      dists.data_ptr<float>(),
      P,
      T);

  AT_CUDA_CHECK(hipGetLastError());
  return dists;
}

__global__ void PointFaceArrayBackwardKernel(
    const float* __restrict__ points, // (P, 3)
    const float* __restrict__ tris, // (T, 3, 3)
    const float* __restrict__ grad_dists, // (P, T)
    float* __restrict__ grad_points, // (P, 3)
    float* __restrict__ grad_tris, // (T, 3, 3)
    const size_t P,
    const size_t T) {
  const float3* points_f3 = (float3*)points;
  const float3* tris_f3 = (float3*)tris;

  // Parallelize over P * S computations
  const int num_threads = gridDim.x * blockDim.x;
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int t_i = tid; t_i < P * T; t_i += num_threads) {
    const int t = t_i / P; // triangle index.
    const int p = t_i % P; // point index
    const float3 v0 = tris_f3[t * 3 + 0];
    const float3 v1 = tris_f3[t * 3 + 1];
    const float3 v2 = tris_f3[t * 3 + 2];

    const float3 point = points_f3[p];

    const float grad_dist = grad_dists[p * T + t];
    const auto grad =
        PointTriangle3DistanceBackward(point, v0, v1, v2, grad_dist);

    const float3 grad_point = thrust::get<0>(grad);
    const float3 grad_v0 = thrust::get<1>(grad);
    const float3 grad_v1 = thrust::get<2>(grad);
    const float3 grad_v2 = thrust::get<3>(grad);

    atomicAdd(grad_points + 3 * p + 0, grad_point.x);
    atomicAdd(grad_points + 3 * p + 1, grad_point.y);
    atomicAdd(grad_points + 3 * p + 2, grad_point.z);

    atomicAdd(grad_tris + t * 3 * 3 + 0 * 3 + 0, grad_v0.x);
    atomicAdd(grad_tris + t * 3 * 3 + 0 * 3 + 1, grad_v0.y);
    atomicAdd(grad_tris + t * 3 * 3 + 0 * 3 + 2, grad_v0.z);

    atomicAdd(grad_tris + t * 3 * 3 + 1 * 3 + 0, grad_v1.x);
    atomicAdd(grad_tris + t * 3 * 3 + 1 * 3 + 1, grad_v1.y);
    atomicAdd(grad_tris + t * 3 * 3 + 1 * 3 + 2, grad_v1.z);

    atomicAdd(grad_tris + t * 3 * 3 + 2 * 3 + 0, grad_v2.x);
    atomicAdd(grad_tris + t * 3 * 3 + 2 * 3 + 1, grad_v2.y);
    atomicAdd(grad_tris + t * 3 * 3 + 2 * 3 + 2, grad_v2.z);
  }
}

std::tuple<at::Tensor, at::Tensor> PointFaceArrayDistanceBackwardCuda(
    const at::Tensor& points,
    const at::Tensor& tris,
    const at::Tensor& grad_dists) {
  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1}, tris_t{tris, "tris", 2},
      grad_dists_t{grad_dists, "grad_dists", 3};
  at::CheckedFrom c = "PointFaceArrayDistanceBackwardCuda";
  at::checkAllSameGPU(c, {points_t, tris_t, grad_dists_t});
  at::checkAllSameType(c, {points_t, tris_t, grad_dists_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int64_t P = points.size(0);
  const int64_t T = tris.size(0);

  TORCH_CHECK(points.size(1) == 3, "points must be of shape Px3");
  TORCH_CHECK(
      (tris.size(1) == 3) && (tris.size(2) == 3),
      "tris must be of shape Tx3x3");
  TORCH_CHECK((grad_dists.size(0) == P) && (grad_dists.size(1) == T));

  at::Tensor grad_points = at::zeros({P, 3}, points.options());
  at::Tensor grad_tris = at::zeros({T, 3, 3}, tris.options());

  if (grad_points.numel() == 0 || grad_tris.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_points, grad_tris);
  }

  const size_t blocks = 1024;
  const size_t threads = 64;

  PointFaceArrayBackwardKernel<<<blocks, threads, 0, stream>>>(
      points.data_ptr<float>(),
      tris.data_ptr<float>(),
      grad_dists.data_ptr<float>(),
      grad_points.data_ptr<float>(),
      grad_tris.data_ptr<float>(),
      P,
      T);

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(grad_points, grad_tris);
}